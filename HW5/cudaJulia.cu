#include "hip/hip_runtime.h"
/* 

To compile:

   gcc -O3 -o mandelbrot mandelbrot.c png_util.c -I. -lpng -lm

To create an image with 4096 x 4096 pixels (last argument will be used to set number of threads):

    ./mandelbrot 4096 4096 1

*/

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "png_util.h"

// Q2a: add include for CUDA header file here:
#include "hip/hip_runtime.h"

#define MXITER 1000


typedef struct {
  
  double r; // real
  double i; // imaginary
  
}complex_t;


// return iterations before z leaves mandelbrot set for given c
__device__ int testpoint(complex_t c, complex_t z){

  int iter;
  double temp;

  z = c;
  
  for(iter = 0; iter < MXITER; iter++){  
    temp = (z.r*z.r) - (z.i*z.i) + c.r;
    
    z.i = z.r*z.i*2. + c.i;
    z.r = temp;
    
    if((z.r*z.r+z.i*z.i) > 4.0){
      return iter;
    }
  }
  return iter; 
}


// perform Julia iteration on a grid of numbers in the complex plane
// record the  iteration counts in the count array
__global__ void julia(int Nre, int Nim, complex_t zmin, complex_t dz, complex_t c, float *count){ 

  // Q2c: replace this loop with a CUDA kernel
  complex_t z;
  int thread = threadIdx.x;
  int block = blockIdx.x;
  int blockSize = blockDim.x;
  int id = block*blockSize + thread;

  int m = id%Nre; // real axis
  int n = id%Nim; // imag axis

  z.r = zmin.r + dz.r*m;
  z.i = zmin.i + dz.i*n;
     
  count[m + n*Nre] = (float) testpoint(c, z);
}


/**
Main method
*/
int main(int argc, char **argv){

  // to create a 4096x4096 pixel image [ last argument is placeholder for number of threads ] 
  // usage: ./mandelbrot 4096 4096 32 

  int Nre = atoi(argv[1]);
  int Nim = atoi(argv[2]);
  int Nthreads = atoi(argv[3]);

  // Q2b: set the number of threads per block and the number of blocks here:
  int Nblocks = (Nre*Nim + Nthreads-1)/Nthreads;

  // storage for the iteration counts
  float *count;
  float *device_count;
  count = (float*)malloc(Nre*Nim*sizeof(float));
  hipMalloc(&device_count, Nre*Nim*sizeof(float));

  // Parameters for a bounding box for "c" that generates an interesting image
  const float centRe = -1.6, centIm = 0.312;
  const float diam = 3.14;

  complex_t zmin; 
  complex_t zmax;
  complex_t dz;
  complex_t c;

  zmin.r = centRe - 0.5*diam;
  zmax.r = centRe + 0.5*diam;
  zmin.i = centIm - 0.5*diam;
  zmax.i = centIm + 0.5*diam;

  //set step sizes
  dz.r = (zmax.r-zmin.r)/(Nre-1);
  dz.i = (zmax.i-zmin.i)/(Nim-1);

  c.i = 0.1560;
  c.r = -0.8;

  clock_t start = clock(); //start time in CPU cycles

  // compute julia set
  julia <<<Nthreads, Nblocks>>> (Nre, Nim, zmin, dz, c, count); 
  
  // copy from the GPU back to the host here
  hipMemcpy(count, device_count, Nre*Nim*sizeof(float), hipMemcpyDeviceToHost);

  clock_t end = clock(); //start time in CPU cycles
  
  // print elapsed time
  printf("elapsed = %f\n", ((double)(end-start))/CLOCKS_PER_SEC);


  // output julia to png format image
  FILE *fp = fopen("julia.png", "w");

  printf("Printing julia.png...");
  write_hot_png(fp, Nre, Nim, count, 0, 80);
  printf("done.\n");

  free(count);

  exit(0);
  return 0;
}  